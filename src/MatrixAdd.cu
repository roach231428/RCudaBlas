#include "hip/hip_runtime.h"
#include <cmath>
#include "cudaFun.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void mat_add(const double* mat1, const double* mat2, double* result, const int M, const int N) {
    int iRow = blockDim.x * blockIdx.x + threadIdx.x;
    int iCol = blockDim.y * blockIdx.y + threadIdx.y;
    int threadId = iRow + iCol * N;
    
    if (threadId < M * N)
        result[threadId] = mat1[threadId] + mat2[threadId];
}


void matrix_add_gpu(const double* mat1, const double* mat2, double* result, const int M, const int N)
{
    double *g_mat1, *g_mat2, *g_mat_result;
    
    hipMalloc((void **)&g_mat1, sizeof(double) * M*N);
    hipMalloc((void **)&g_mat2, sizeof(double) * M*N);
    hipMalloc((void **)&g_mat_result, sizeof(double) * M*N);

    hipMemcpy(g_mat1, mat1, sizeof(double) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(g_mat2, mat2, sizeof(double) * M*N, hipMemcpyHostToDevice);
    
    dim3 blockSize(32, 32);
    dim3 gridSize(20, 20);
    
    mat_add<<< gridSize, blockSize >>>(g_mat1, g_mat2, g_mat_result, M, N);

    hipMemcpy(result, g_mat_result, sizeof(double) * M*N, hipMemcpyDeviceToHost);
    
    hipFree(g_mat1);
    hipFree(g_mat2);
    hipFree(g_mat_result);
}