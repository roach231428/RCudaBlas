#include "hip/hip_runtime.h"
#include <cmath>
#include "cudaFun.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


__global__ void mat_mul(const double* mat1, const double* mat2, double* result, const int M, const int N, const int S) {
    int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
                    + blockIdx.x * blockDim.x + threadIdx.x;
    
    if (threadId < M * S)
    {
        int row = threadId / M;
        int column = threadId % M;
        
        result[threadId] = 0;
        for (int j = 0; j < N; j++)
        {
            // result[threadId] += mat1[row * N + j] * mat2[j * S + column];
            result[threadId] += mat1[j * M + column] * mat2[row * N + j];
        }
    }
}


void matrix_multiplication_gpu(const double* mat1, const double* mat2, double* result, const int M, const int N, const int S)
{
    double *g_mat1, *g_mat2, *g_mat_result;
    
    hipMalloc((void **)&g_mat1, sizeof(double) * M*N);
    hipMalloc((void **)&g_mat2, sizeof(double) * N*S);
    hipMalloc((void **)&g_mat_result, sizeof(double) * M*S);

    hipMemcpy(g_mat1, mat1, sizeof(double) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(g_mat2, mat2, sizeof(double) * N*S, hipMemcpyHostToDevice);
    
    dim3 blockSize(32, 32);
    dim3 gridSize(20, 20);
    
    mat_mul<<< gridSize, blockSize >>>(g_mat1, g_mat2, g_mat_result, M, N, S);

    hipMemcpy(result, g_mat_result, sizeof(double) * M*S, hipMemcpyDeviceToHost);
    
    hipFree(g_mat1);
    hipFree(g_mat2);
    hipFree(g_mat_result);
}